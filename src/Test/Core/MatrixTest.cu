#include "hip/hip_runtime.h"
#include "Test/Core/MatrixTest.h"
#include <iostream>
#include "Core/Timer/Timer.h"
#include <random>
#include "Core/LinearAlgebra/gemm.h"

using namespace Polaris;

HOST_FUNC Matrix<Scalar, Platform::CPU> GetRandomMatrix(Label m, Label n, PlsTimer& timer)
{
	timer.HostStart("construct matrix");
	Matrix<Scalar, Platform::CPU> matrix(m, n);
	static std::random_device rd; 
	static std::mt19937 gen(rd()); 
	std::uniform_real_distribution<Scalar> dis(0.0, 1.0);
	PLS_FOR_I(m)
	{
		PLS_FOR_J(n)
		{
			matrix(i, j) = dis(gen);
		}
	}
	timer.HostStop("construct matrix");
	return matrix;
}

void Polaris::Device::MatrixTest()
{
	//static constexpr size_t M = 2048;
	//static constexpr size_t N = 2048;

	static constexpr size_t M = 1024;
	static constexpr size_t N = 1024;
	static constexpr size_t K = 1024;

	auto& timer = PlsTimer::GetInstance();

	auto matrix1 = GetRandomMatrix(M, N, timer);
	auto matrix2 = GetRandomMatrix(N, K, timer);

	Matrix<Scalar, Platform::CUDA> gmatrix1;
	gmatrix1.Transfer(matrix1);
	Matrix<Scalar, Platform::CUDA> gmatrix2;
	gmatrix2.Transfer(matrix2);

	sgemm<Platform::CPU, std::allocator<Scalar>> gemm;
	auto& cpuRes = gemm(matrix1, matrix2, &timer);
	sgemm<Platform::CUDA, cuda_utils::CudaAllocator<Scalar>> ggemm;
	auto& gpuRes = ggemm(gmatrix1, gmatrix2, &timer);

	Matrix<Scalar, Platform::CPU> gpuResToCpu;
	gpuResToCpu.Transfer(gpuRes);

	Scalar eps = 1e-10f;
	bool isSame = true;
	PLS_FOR_I(M)
	{
		PLS_FOR_J(K)
		{
			bool elementTest = abs(cpuRes(i, j) - gpuResToCpu(i, j)) < eps;
			if (!elementTest)
			{
				PLS_WARN("{}, {} element dismatch: {}-{},{}", i, j, cpuRes(i, j), gpuResToCpu(i, j),abs(cpuRes(i, j) - gpuResToCpu(i, j)));
			}
			isSame = isSame && elementTest;
		}
	}
	if (isSame)
	{
		PLS_INFO("gemm test finish");
	}

	timer.PrintAll();
}